// matrix_mul.cu
#include <hip/hip_runtime.h>

// CUDA kernel for matrix multiplication
__global__ void matrix_mul_cuda(float* A, float* B, float* C, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < N && col < N) {
        float sum = 0.0f;
        for (int k = 0; k < N; k++) {
            sum += A[row * N + k] * B[k * N + col];
        }
        C[row * N + col] = sum;
    }
}

// Function to call the CUDA kernel
extern "C" void call_cuda_kernel(float* A, float* B, float* C, int N) {
    size_t size = N * N * sizeof(float);

    // Allocate device memory
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    // Copy data to device
    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    // Define block and grid dimensions
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((N + threadsPerBlock.x - 1) / threadsPerBlock.x,
                      (N + threadsPerBlock.y - 1) / threadsPerBlock.y);

    // Launch the CUDA kernel
    matrix_mul_cuda<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

    // Copy result back to host
    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}